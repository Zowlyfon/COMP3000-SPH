#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""

#include "utility.cuh"

// float3 operators

__device__ float3 operator-(const float3& a, const float3& b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 operator+(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator/(const float3& a, const float b) {
    return make_float3(a.x / b, a.y / b, a.z / b);
}

__device__ float3 operator/(const float a, const float3& b) {
    return make_float3(a / b.x, a / b.y, a / b.z);
}

__device__ float3 operator*(const float3& a, const float b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ float3 operator*(const float b, const float3& a) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ int CalculateParticleCellID(Parameters Params, float3 Pos) {
    int numXCell = (Params.DomainX / Params.H) + 1;
    int numYCell = (Params.DomainY / Params.H) + 1;
    int numZCell = (Params.DomainZ / Params.H) + 1;

    int xCell = Pos.x / Params.H;
    int yCell = Pos.y / Params.H;
    int zCell = Pos.z / Params.H;

    return (((xCell * numYCell) + yCell) * numXCell) + zCell;
}

__device__ float Sqr(float x) {
    return x * x;
}

__device__ float Sqr(float3 v) {
    return v.x * v.x + v.y * v.y + v.z * v.z;
}