#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "utility.cuh"

// float3 operators

__device__ float3 operator-(const float3& a, const float3& b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 operator+(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator/(const float3& a, const float b) {
    return make_float3(a.x / b, a.y / b, a.z / b);
}

__device__ float3 operator/(const float a, const float3& b) {
    return make_float3(a / b.x, a / b.y, a / b.z);
}

__device__ float3 operator*(const float3& a, const float b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ float3 operator*(const float b, const float3& a) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}