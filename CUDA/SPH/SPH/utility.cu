#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""

#include "utility.cuh"

// float3 operators

__host__ __device__ float3 operator-(const float3& a, const float3& b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__host__ __device__ float3 operator+(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ __device__ float3 operator+(const float3& a, const float& b) {
    return make_float3(a.x + b, a.y + b, a.z + b);
}

__host__ __device__ float3 operator/(const float3& a, const float b) {
    return make_float3(a.x / b, a.y / b, a.z / b);
}

__host__ __device__ float3 operator/(const float a, const float3& b) {
    return make_float3(a / b.x, a / b.y, a / b.z);
}

__host__ __device__ float3 operator*(const float3& a, const float b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__host__ __device__ float3 operator*(const float b, const float3& a) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__host__ __device__ float3 operator*(const float3 a, const float3 b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__host__ __device__ float Dot(float3 a, float3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ float Sqr(float x) {
    return x * x;
}

__host__ __device__ float Sqr(float3 v) {
    return v.x * v.x + v.y * v.y + v.z * v.z;
}

__host__ __device__ float3 Abs(float3 a) {
    return make_float3(fabs(a.x), fabs(a.y), fabs(a.z));
}

__host__ __device__ float Length(float3 a) {
    float R2 = Sqr(a);
    return sqrtf(R2);
}

__host__ __device__ float3 Normalise(float3 a) {
    float R = Length(a);
    return a / R;
}

__host__ __device__ float3 Max(float3 a, float3 b) {
    return make_float3(fmax(a.x, b.x), fmax(a.y, b.y), fmax(a.z, b.z));
}

__host__ __device__ float3 Min(float3 a, float3 b) {
    return make_float3(fmin(a.x, b.x), fmin(a.y, b.y), fmin(a.z, b.z));
}

__host__ __device__ int CalculateParticleCellID(Parameters Params, float3 Pos) {
    int numXCell = (Params.Domain.x / Params.H) + 1;
    int numYCell = (Params.Domain.y / Params.H) + 1;
    int numZCell = (Params.Domain.z / Params.H) + 1;

    int xCell = Pos.x / Params.H;
    int yCell = Pos.y / Params.H;
    int zCell = Pos.z / Params.H;

    return (((xCell * numYCell) + yCell) * numZCell) + zCell;
}


__host__ __device__ float3 BoxToSDF(float3 BoxStart, float3 BoxEnd) {
    return Abs(BoxStart) + BoxEnd;
}

// https://iquilezles.org/www/articles/distfunctions/distfunctions.htm
__host__ __device__ float BoxSDF(float3 X, float3 B, float r) {
    float3 Q = Abs(X) - (B / 2.0f);
    return Length(Max(Q, make_float3(0.0f, 0.0f, 0.0f))) + fmin(fmax(Q.x, fmax(Q.y, Q.z)), 0.0f) - r;
}

// https://www.iquilezles.org/www/articles/normalsSDF/normalsSDF.htm
__host__ __device__ float3 CalculateBoxSDFNormal(float3 X, float3 B, float r) {
    const float2 R = make_float2(0.0001, 0);
    return Normalise(make_float3(
        BoxSDF(X + make_float3(R.x, R.y, R.y), B, r) - BoxSDF(X - make_float3(R.x, R.y, R.y), B, r),
        BoxSDF(X + make_float3(R.y, R.x, R.y), B, r) - BoxSDF(X - make_float3(R.y, R.x, R.y), B, r),
        BoxSDF(X + make_float3(R.y, R.y, R.x), B, r) - BoxSDF(X - make_float3(R.y, R.y, R.x), B, r)));
}