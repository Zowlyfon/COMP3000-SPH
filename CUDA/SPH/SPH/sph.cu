#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>

#include "sph.cuh"
#include "utility.cuh"

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define CudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ float Poly6(const float H) {
	return 315.0 / (64.0f * M_PI * pow(H, 9.f));
}

__device__ float SpikyGrad(const float H) {
	return -15.f / (M_PI * pow(H, 6.f));
}

__device__ float ViscLap(const float H) {
	return 45.f / (M_PI * pow(H, 6.f));
}

__global__ void CalculateNeighbours(Parameters Params, Particles P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P.NumP) {
		int NumXCell = (Params.Domain.x / Params.H) + 1;
		int NumYCell = (Params.Domain.y / Params.H) + 1;
		int NumZCell = (Params.Domain.z / Params.H) + 1;

		int XCell = P.X[i].x / Params.H;
		int YCell = P.X[i].y / Params.H;
		int ZCell = P.X[i].z / Params.H;

		int Count = 0;

		for (int x = XCell - 1; x <= XCell + 1; x++) {
			for (int y = YCell - 1; y <= YCell + 1; y++) {
				for (int z = ZCell - 1; z <= ZCell + 1; z++) {
					int id = (((x * NumYCell) + y) * NumZCell) + z;
					P.NeighbourCells[(i * 27) + Count] = (x >= 0 && x < NumXCell&& y >= 0 && y < NumYCell&& z >= 0 && z < NumZCell) ? id : -1;
					Count++;
				}
			}
		}
	}
}

__global__ void ComputeDensityPressure(Parameters Params, Particles P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P.NumP) {

		// Compute Density Pressure
		float Rho = 0.0f;

		for (int k = 0; k < 27; k++) {
			// Cell is between 0 and NumCells
			if (!(P.NeighbourCells[i * 27 + k] < P.NumCells && P.NeighbourCells[i * 27 + k] >= 0)) {
				continue;
			}


			int CellStart = P.Cells[P.NeighbourCells[i * 27 + k] * 2];
			int CellEnd = CellStart + P.Cells[P.NeighbourCells[i * 27 + k] * 2 + 1] - 1;

			// Number of particles in cell is atleast 1
			if (CellEnd < CellStart) {
				continue;
			}

			for (int j = CellStart; j <= CellEnd; j++) {
				float3 Rij;

				// Distance between particle i and j
				Rij = P.X[j] - P.X[i];

				// Squared distance
				float R2 = Sqr(Rij);

				// particle j is within the smoothing radius H
				if (R2 < Sqr(Params.H)) {
					Rho += Params.Mass * Poly6(Params.H) * powf(Sqr(Params.H) - R2, 3.0f);
				}
			}
		}

		P.Rho[i] = Rho;
		P.P[i] = Params.GasConst * (Rho - Params.RestDensity);
	}
}

__global__ void ComputeForces(Parameters Params, Particles P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P.NumP) {
		float3 FPress = make_float3(0.0f, 0.0f, 0.0f);
		float3 FVisc = make_float3(0.0f, 0.0f, 0.0f);

		for (int k = 0; k < 27; k++) {
			// Cell is between 0 and NumCells
			if (!(P.NeighbourCells[i * 27 + k] < P.NumCells && P.NeighbourCells[i * 27 + k] >= 0)) {
				continue;
			}


			int CellStart = P.Cells[P.NeighbourCells[i * 27 + k] * 2];
			int CellEnd = CellStart + P.Cells[P.NeighbourCells[i * 27 + k] * 2 + 1] - 1;

			// Number of particles in cell is atleast 1
			if (CellEnd < CellStart) {
				continue;
			}

			for (int j = CellStart; j <= CellEnd; j++) {
				if (i == j) {
					continue;
				}

				float3 Rij;
				Rij = P.X[j] - P.X[i];
				float R2 = Sqr(Rij);
				float R = sqrtf(R2);

				if (R < Params.H) {
					float3 RNorm = Rij / R;

					float Pij = P.P[i] + P.P[j];

					FPress = FPress + ((RNorm * -1.0f) * Params.Mass * Pij / (2.0f * P.Rho[j]) * SpikyGrad(Params.H) * powf(Params.H - R, 3.0f));

					float3 Vij = P.V[j] - P.V[i];

					FVisc = FVisc + (Params.Visc * Params.Mass * Vij / P.Rho[j] * ViscLap(Params.H) * (Params.H - R));
				}
			}
		}

		float3 FGrav = Params.G * Params.Mass / P.Rho[i];

		P.F[i] = FPress + FVisc + FGrav;
	}
}

__global__ void UpdateVelocityHalf(Parameters Params, Particles P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P.NumP) {
		P.V[i] = P.V[i] + ((Params.DT / 2.0f) * P.F[i] / P.Rho[i]);
	}
}

__global__ void UpdatePosition(Parameters Params, Particles P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P.NumP) {
		P.X[i] = P.X[i] + (Params.DT * P.V[i]);
	}
}

__global__ void ComputeBoundries(Parameters Params, Particles P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P.NumP) {
		if (P.X[i].x - Params.H < 0.0f) {
			P.V[i].x = P.V[i].x * Params.BoundDamp;
			P.X[i].x = Params.H;
		}
		if (P.X[i].x + Params.H > Params.Domain.x) {
			P.V[i].x = P.V[i].x * Params.BoundDamp;
			P.X[i].x = Params.Domain.x - Params.H;
		}

		if (P.X[i].y - Params.H < 0.0f) {
			P.V[i].y = P.V[i].y * Params.BoundDamp;
			P.X[i].y = Params.H;
		}
		if (P.X[i].y + Params.H > Params.Domain.y) {
			P.V[i].y = P.V[i].y * Params.BoundDamp;
			P.X[i].y = Params.Domain.y - Params.H;
		}

		if (P.X[i].z - Params.H < 0.0f) {
			P.V[i].z = P.V[i].z * Params.BoundDamp;
			P.X[i].z = Params.H;
		}
		if (P.X[i].z + Params.H > Params.Domain.z) {
			P.V[i].z = P.V[i].z * Params.BoundDamp;
			P.X[i].z = Params.Domain.z - Params.H;
		}
	}
}


/*
__global__ void ComputeCollisions(Parameters Params, Particles* P, float3 ColliderStart, float3 ColliderEnd) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P.NumP) {
		if (P.X[i].x >= ColliderStart.x && P.X[i].x <= ColliderEnd.x &&
			P.X[i].y >= ColliderStart.y && P.X[i].y <= ColliderEnd.y &&
			P.X[i].z >= ColliderStart.z && P.X[i].z <= ColliderEnd.z) {
			float RX = P.X[i].x - ColliderStart.x;
			float NRX = ColliderEnd.x - P.X[i].x;

			float RY = P.X[i].y - ColliderStart.y;
			float NRY = ColliderEnd.y - P.X[i].y;

			float RZ = P.X[i].z - ColliderStart.z;
			float NRZ = ColliderEnd.z - P.X[i].z;

			if (RX < NRX && RX < RY && RX < NRY && RX < RZ && RX < NRZ) {
				P.X 
			}
		}
	}
}
*/

__global__ void ComputeCollisions(Parameters Params, Particles P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P.NumP) {
		float3 NX = P.X[i] - Params.CollisionBoxTransform;
		float Distance = BoxSDF(NX, Params.CollisionBox, Params.H / 2.0f);
		if (Distance < 0.0f) {
#ifndef NDEBUG
			float3 X = P.X[i];
			float3 V = P.V[i];
#endif
			/*
			float3 R = (Params.CollisionBoxEnd - Params.CollisionBoxStart) / 2.0f;
			float3 BoxStart = -1.0f * R;
			//float3 BoxStart = Params.CollisionBoxStart - (Params.CollisionBoxStart + R);
			float3 BoxEnd = R;
			//float3 BoxEnd = Params.CollisionBoxEnd - (Params.CollisionBoxStart + R);
			float3 Box = BoxToSDF(BoxStart, BoxEnd);
			*/
			
			float3 Normal = CalculateBoxSDFNormal(NX, Params.CollisionBox, Params.H / 2.0f);
#ifndef NDEBUG
			float3 NormDist = Normal * Distance;
#endif
			P.X[i] = P.X[i] + Normal * Distance * -1.0f;
			float3 DampFactor = Normal * (Params.BoundDamp * -1.0f);
			//P.V[i] = P.V[i] - (P.V[i] * Normal * (Params.BoundDamp * -1.0f));
			P.V[i] = P.V[i] - 2 * Dot(P.V[i], Normal) * (Normal * 0.75);
#ifndef NDEBUG
			float3 UV = P.V[i];
			float3 UX = P.X[i];
			if (Normal.y >= 0.99 || Normal.y <= -0.99) {
				UV = P.V[i];
			}
#endif
			//P.V[i] = P.V[i] * Params.BoundDamp;
		}
	}
}

__global__ void FindParticleCellIDs(Parameters Params, Particles P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < P.NumP) {
		int CellID = CalculateParticleCellID(Params, P.X[i]);
		P.ParticleCellID[i] = CellID;
	}
}

__global__ void SetIndices(int* Indices, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		Indices[i] = i;
	}
}

__global__ void SortParticleData(int* Indices, Particles P, float3* TX, float3* TV, float3* TF, float* TP, float* TRho) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < P.NumP) {
		TX[i] = P.X[Indices[i]];
		TV[i] = P.V[Indices[i]];
		TF[i] = P.F[Indices[i]];
		TP[i] = P.P[Indices[i]];
		TRho[i] = P.Rho[Indices[i]];
	}
}

__global__ void CreateCells(int* Histogram, int* PrefixSum, int* Cells, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		Cells[i * 2] = PrefixSum[i];
		Cells[i * 2 + 1] = Histogram[i];
	}
}

void PutParticlesInCells(Parameters Params, Particles P) {
	int BlockSize = 256;
	int NumBlocks = (P.NumP + BlockSize - 1) / BlockSize;

	FindParticleCellIDs << <NumBlocks, BlockSize >> > (Params, P);
	//hipDeviceSynchronize();

	int* ParticleCellID_Out;
	int* Indices_In;
	int* Indices_Out;

	CudaErrorCheck(hipMalloc(&ParticleCellID_Out, sizeof(int) * P.NumP));
	CudaErrorCheck(hipMalloc(&Indices_In, sizeof(int) * P.NumP));
	CudaErrorCheck(hipMalloc(&Indices_Out, sizeof(int) * P.NumP));

	SetIndices << <NumBlocks, BlockSize >> > (Indices_In, P.NumP);

	//hipDeviceSynchronize();

	void* TempStorage = NULL;
	size_t TempStorageBytes = 0;

	hipcub::DeviceRadixSort::SortPairs(TempStorage, TempStorageBytes, P.ParticleCellID, ParticleCellID_Out, Indices_In, Indices_Out, P.NumP);

	CudaErrorCheck(hipMalloc(&TempStorage, TempStorageBytes));

	hipcub::DeviceRadixSort::SortPairs(TempStorage, TempStorageBytes, P.ParticleCellID, ParticleCellID_Out, Indices_In, Indices_Out, P.NumP);

	//hipDeviceSynchronize();
	
	CudaErrorCheck(hipFree(TempStorage));

	CudaErrorCheck(hipMemcpy(P.ParticleCellID, ParticleCellID_Out, sizeof(int) * P.NumP, hipMemcpyDeviceToDevice));
	
	CudaErrorCheck(hipFree(ParticleCellID_Out));
	CudaErrorCheck(hipFree(Indices_In));

	float3* TX;
	float3* TV;
	float3* TF;
	float* TP;
	float* TRho;

	CudaErrorCheck(hipMalloc(&TX, sizeof(float3) * P.NumP));
	CudaErrorCheck(hipMalloc(&TV, sizeof(float3) * P.NumP));
	CudaErrorCheck(hipMalloc(&TF, sizeof(float3) * P.NumP));
	CudaErrorCheck(hipMalloc(&TP, sizeof(float) * P.NumP));
	CudaErrorCheck(hipMalloc(&TRho, sizeof(float) * P.NumP));

	SortParticleData << <NumBlocks, BlockSize >> > (Indices_Out, P, TX, TV, TF, TP, TRho);
	//hipDeviceSynchronize();

	CudaErrorCheck(hipMemcpy(P.X, TX, sizeof(float3) * P.NumP, hipMemcpyDeviceToDevice));
	CudaErrorCheck(hipMemcpy(P.V, TV, sizeof(float3) * P.NumP, hipMemcpyDeviceToDevice));
	CudaErrorCheck(hipMemcpy(P.F, TF, sizeof(float3) * P.NumP, hipMemcpyDeviceToDevice));
	CudaErrorCheck(hipMemcpy(P.P, TP, sizeof(float) * P.NumP, hipMemcpyDeviceToDevice));
	CudaErrorCheck(hipMemcpy(P.Rho, TRho, sizeof(float) * P.NumP, hipMemcpyDeviceToDevice));

	CudaErrorCheck(hipFree(TX));
	CudaErrorCheck(hipFree(TV));
	CudaErrorCheck(hipFree(TF));
	CudaErrorCheck(hipFree(TP));
	CudaErrorCheck(hipFree(TRho));

	CudaErrorCheck(hipFree(Indices_Out));

	int* Histogram;
	CudaErrorCheck(hipMalloc(&Histogram, sizeof(int) * P.NumCells));

	TempStorage = NULL;
	TempStorageBytes = 0;

	hipcub::DeviceHistogram::HistogramEven(TempStorage, TempStorageBytes, P.ParticleCellID, Histogram, P.NumCells, 0, P.NumCells, P.NumP);

	CudaErrorCheck(hipMalloc(&TempStorage, TempStorageBytes));

	hipcub::DeviceHistogram::HistogramEven(TempStorage, TempStorageBytes, P.ParticleCellID, Histogram, P.NumCells, 0, P.NumCells, P.NumP);

	//hipDeviceSynchronize();

	CudaErrorCheck(hipFree(TempStorage));

	int* PrefixSum;

	CudaErrorCheck(hipMalloc(&PrefixSum, sizeof(int) * P.NumCells));

	TempStorage = NULL;
	TempStorageBytes = 0;

	hipcub::DeviceScan::ExclusiveSum(TempStorage, TempStorageBytes, Histogram, PrefixSum, P.NumCells);

	CudaErrorCheck(hipMalloc(&TempStorage, TempStorageBytes));

	hipcub::DeviceScan::ExclusiveSum(TempStorage, TempStorageBytes, Histogram, PrefixSum, P.NumCells);
	//hipDeviceSynchronize();

	CudaErrorCheck(hipFree(TempStorage));

	NumBlocks = (P.NumCells + BlockSize - 1) / BlockSize;

	CreateCells << <NumBlocks, BlockSize >> > (Histogram, PrefixSum, P.Cells, P.NumCells);
	//hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));
	}

	CudaErrorCheck(hipFree(Histogram));
	CudaErrorCheck(hipFree(PrefixSum));
}

Particles AllocateMemory(int NumP, int NumCells) {
	//Particles* P;
	//hipMallocManaged(&P, sizeof(Particles));
	Particles P;
	CudaErrorCheck(hipMallocManaged(&(P.X), sizeof(float3) * NumP));
	CudaErrorCheck(hipMallocManaged(&(P.F), sizeof(float3) * NumP));
	CudaErrorCheck(hipMallocManaged(&(P.V), sizeof(float3) * NumP));
	CudaErrorCheck(hipMallocManaged(&(P.P), sizeof(float) * NumP));
	CudaErrorCheck(hipMallocManaged(&(P.Rho), sizeof(float) * NumP));
	CudaErrorCheck(hipMallocManaged(&(P.Cells), sizeof(int) * NumCells * 2));
	CudaErrorCheck(hipMallocManaged(&(P.NeighbourCells), sizeof(int) * NumP * 27));
	CudaErrorCheck(hipMallocManaged(&(P.ParticleCellID), sizeof(float) * NumP));

	P.NumP = NumP;
	P.NumCells = NumCells;

	return P;
}

void FreeMemory(Particles P) {
	CudaErrorCheck(hipFree(P.ParticleCellID));
	CudaErrorCheck(hipFree(P.NeighbourCells));
	CudaErrorCheck(hipFree(P.Cells));
	CudaErrorCheck(hipFree(P.Rho));
	CudaErrorCheck(hipFree(P.P));
	CudaErrorCheck(hipFree(P.V));
	CudaErrorCheck(hipFree(P.F));
	CudaErrorCheck(hipFree(P.X));
}

void InitSph(Parameters Params, Particles P, float3 Start, float3 End) {
	for (int i = 0; i < P.NumP; i++) {
		P.V[i] = make_float3(0.0f, 0.0f, 0.0f);
		P.F[i] = make_float3(0.0f, 0.0f, 0.0f);
		P.P[i] = 0.0f;
		P.Rho[i] = 0.0f;
	}

	for (int i = 0; i < P.NumCells; i++) {
		P.Cells[i] = -1;
	}

	int nParticles = 0;

	for (float x = Start.x; x <= End.x; x += Params.H) {
		for (float y = Start.y; y <= End.y; y += Params.H) {
			for (float z = Start.z; z <= End.z; z += Params.H) {
				if (nParticles < P.NumP) {
					float jitter = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
					P.X[nParticles].x = x + jitter;
					P.X[nParticles].y = y + jitter;
					P.X[nParticles].z = z;
					nParticles++;
				}
				else {
					return;
				}
			}
		}
	}
}

void FirstStep(Parameters Params, Particles P) {
	int BlockSize = 256;
	int NumBlocks = (P.NumP + BlockSize - 1) / BlockSize;

	CalculateNeighbours << <NumBlocks, BlockSize >> > (Params, P);
	ComputeDensityPressure << <NumBlocks, BlockSize >> > (Params, P);
	ComputeForces << <NumBlocks, BlockSize >> > (Params, P);
	UpdateVelocityHalf << <NumBlocks, BlockSize >> > (Params, P);
	//UpdateVelocityHalf << <NumBlocks, BlockSize >> > (Params, P);
	UpdatePosition << <NumBlocks, BlockSize >> > (Params, P);
	ComputeBoundries << <NumBlocks, BlockSize >> > (Params, P);
	if (Params.DoCollisions) {
		ComputeCollisions << <NumBlocks, BlockSize >> > (Params, P);
	}
	CudaErrorCheck(hipDeviceSynchronize());
}

void SimStep(Parameters Params, Particles P) {
	int BlockSize = 256;
	int NumBlocks = (P.NumP + BlockSize - 1) / BlockSize;

	CalculateNeighbours << <NumBlocks, BlockSize >> > (Params, P);
	ComputeDensityPressure << <NumBlocks, BlockSize >> > (Params, P);
	ComputeForces << <NumBlocks, BlockSize >> > (Params, P);
	UpdateVelocityHalf << <NumBlocks, BlockSize >> > (Params, P);
	UpdateVelocityHalf << <NumBlocks, BlockSize >> > (Params, P);
	UpdatePosition << <NumBlocks, BlockSize >> > (Params, P);
	ComputeBoundries << <NumBlocks, BlockSize >> > (Params, P);
	if (Params.DoCollisions) {
		ComputeCollisions << <NumBlocks, BlockSize >> > (Params, P);
	}
	//CudaErrorCheck(hipDeviceSynchronize());
}

#ifndef NDEBUG
int main() {
	Parameters Params;
	Params.Domain = make_float3(500.0, 500.0, 500.0);
	float3 SpawnBoxStart = make_float3((Params.Domain.x / 2) - (Params.Domain.x / 4), (Params.Domain.y / 2) - (Params.Domain.y / 4), (Params.Domain.z / 2) - (Params.Domain.z / 4));
	float3 SpawnBoxEnd = make_float3((Params.Domain.x / 2) + (Params.Domain.x / 4), (Params.Domain.y / 2) + (Params.Domain.y / 4), (Params.Domain.z / 2) + (Params.Domain.z / 4));
	int numP = ((SpawnBoxEnd.x - SpawnBoxStart.x) / Params.H) * ((SpawnBoxEnd.y - SpawnBoxStart.y) / Params.H) * ((SpawnBoxEnd.z - SpawnBoxStart.z) / Params.H);
	int numCells = ((Params.Domain.x / Params.H) + 1) * ((Params.Domain.y / Params.H) + 1) * ((Params.Domain.z / Params.H) + 1);
	Particles P = AllocateMemory(numP, numCells);
	bool ShouldRun = true;

	Params.CollisionBoxStart = make_float3(-1.0f, 70.f, -1.0f);
	Params.CollisionBoxEnd = make_float3(101.0f, 100.0f, 15.0f);
	float3 R = (Params.CollisionBoxEnd - Params.CollisionBoxStart) / 2.0f;
	Params.CollisionBox = BoxToSDF(-1.0f * R, R);
	Params.CollisionBoxTransform = Params.CollisionBoxStart + R;
	Params.DoCollisions = true;


	InitSph(Params,
		P,
		SpawnBoxStart,
		SpawnBoxEnd);
	PutParticlesInCells(Params, P);
	CudaErrorCheck(hipDeviceSynchronize());
	FirstStep(Params, P);

	int steps = 0;
	while (ShouldRun) {
		PutParticlesInCells(Params, P);
		SimStep(Params, P);
		CudaErrorCheck(hipDeviceSynchronize());
		//steps++;
		if (steps >= 2) {
			ShouldRun = false;
		}
	}

	FreeMemory(P);
}
#endif