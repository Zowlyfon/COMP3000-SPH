#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>

#include "sph.cuh"
#include "utility.cuh"

__device__ float Poly6(const float H) {
	return 315.0 / (64.0f * M_PI * pow(H, 9.f));
}

__device__ float SpikyGrad(const float H) {
	return -15.f / (M_PI * pow(H, 6.f));
}

__device__ float ViscLap(const float H) {
	return 45.f / (M_PI * pow(H, 6.f));
}

__global__ void ComputeDensityPressure(Parameters Params, Particles* P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P->NumP) {

		// Compute Density Pressure
		float Rho = 0.0f;

		for (int k = 0; k < 27; k++) {
			// Cell is between 0 and NumCells
			if ((P->NeighbourCells[i * 27 + k] < P->NumCells && P->NeighbourCells[i * 27 + k] >= 0)) {
				continue;
			}


			int CellStart = P->Cells[P->NeighbourCells[i * 27 + k] * 2];
			int CellEnd = CellStart + P->Cells[P->NeighbourCells[i * 27 + k] * 2 + 1] - 1;

			// Number of particles in cell is atleast 1
			if (CellEnd < CellStart) {
				continue;
			}

			for (int j = CellStart; j <= CellEnd; j++) {
				float3 Rij;

				// Distance between particle i and j
				Rij = P->X[j] - P->X[i];

				// Squared distance
				float R2 = Sqr(Rij);

				// particle j is within the smoothing radius H
				if (R2 < Sqr(Params.H)) {
					Rho += Params.Mass * Poly6(Params.H) * powf(Sqr(Params.H) - R2, 3.0f);
				}
			}
		}

		P->Rho[i] = Rho;
		P->P[i] = Params.GasConst * (Rho - Params.RestDensity);
	}
}

__global__ void ComputeForces(Parameters Params, Particles* P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P->NumP) {
		float3 FPress = make_float3(0.0f, 0.0f, 0.0f);
		float3 FVisc = make_float3(0.0f, 0.0f, 0.0f);

		for (int k = 0; k < 27; k++) {
			// Cell is between 0 and NumCells
			if ((P->NeighbourCells[i * 27 + k] < P->NumCells && P->NeighbourCells[i * 27 + k] >= 0)) {
				continue;
			}


			int CellStart = P->Cells[P->NeighbourCells[i * 27 + k] * 2];
			int CellEnd = CellStart + P->Cells[P->NeighbourCells[i * 27 + k] * 2 + 1] - 1;

			// Number of particles in cell is atleast 1
			if (CellEnd < CellStart) {
				continue;
			}

			for (int j = CellStart; j <= CellEnd; j++) {
				if (i == j) {
					continue;
				}

				float3 Rij;
				Rij = P->X[j] - P->X[i];
				float R2 = Sqr(Rij);
				float R = sqrtf(R2);

				if (R < Params.H) {
					float3 RNorm = Rij / R;

					float Pij = P->P[i] + P->P[j];

					FPress = FPress + ((RNorm * -1.0f) * Params.Mass * Pij / (2.0f * P->Rho[j]) * SpikyGrad(Params.H) * powf(Params.H - R, 3.0f));

					float3 Vij = P->V[j] - P->V[i];

					FVisc = FVisc + (Params.Visc * Params.Mass * Vij / P->Rho[j] * ViscLap(Params.H) * (Params.H - R));
				}
			}
		}

		float3 FGrav = Params.G * Params.Mass / P->Rho[i];

		P->F[i] = FPress + FVisc + FGrav;
	}
}

__global__ void UpdateVelocityHalf(Parameters Params, Particles* P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P->NumP) {
		P->V[i] = P->V[i] + ((Params.DT / 2.0f) * P->F[i] / P->Rho[i]);
	}
}