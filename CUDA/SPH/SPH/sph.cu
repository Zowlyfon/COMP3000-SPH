#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>

#include "sph.cuh"
#include "utility.cuh"

__device__ float Poly6(const float H) {
	return 315.0 / (64.0f * M_PI * pow(H, 9.f));
}

__device__ float SpikyGrad(const float H) {
	return -15.f / (M_PI * pow(H, 6.f));
}

__device__ float ViscLap(const float H) {
	return 45.f / (M_PI * pow(H, 6.f));
}

__global__ void CalculateNeighbours(Parameters Params, Particles* P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P->NumP) {
		int NumXCell = (Params.DomainX / Params.H) + 1;
		int NumYCell = (Params.DomainY / Params.H) + 1;
		int NumZCell = (Params.DomainZ / Params.H) + 1;

		int XCell = P->X->x / Params.H;
		int YCell = P->X->y / Params.H;
		int ZCell = P->X->z / Params.H;

		int Count = 0;

		for (int x = XCell - 1; x <= XCell + 1; x++) {
			for (int y = YCell - 1; y <= YCell + 1; y++) {
				for (int z = ZCell - 1; z <= ZCell + 1; z++) {
					int id = (((x * NumYCell) + y) * NumXCell) + z;
					P->NeighbourCells[(i * 27) + Count] = (x >= 0 && x < NumXCell&& y >= 0 && y < NumYCell&& z >= 0 && z < NumZCell) ? id : -1;
					Count++;
				}
			}
		}
	}
}

__global__ void ComputeDensityPressure(Parameters Params, Particles* P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P->NumP) {

		// Compute Density Pressure
		float Rho = 0.0f;

		for (int k = 0; k < 27; k++) {
			// Cell is between 0 and NumCells
			if ((P->NeighbourCells[i * 27 + k] < P->NumCells && P->NeighbourCells[i * 27 + k] >= 0)) {
				continue;
			}


			int CellStart = P->Cells[P->NeighbourCells[i * 27 + k] * 2];
			int CellEnd = CellStart + P->Cells[P->NeighbourCells[i * 27 + k] * 2 + 1] - 1;

			// Number of particles in cell is atleast 1
			if (CellEnd < CellStart) {
				continue;
			}

			for (int j = CellStart; j <= CellEnd; j++) {
				float3 Rij;

				// Distance between particle i and j
				Rij = P->X[j] - P->X[i];

				// Squared distance
				float R2 = Sqr(Rij);

				// particle j is within the smoothing radius H
				if (R2 < Sqr(Params.H)) {
					Rho += Params.Mass * Poly6(Params.H) * powf(Sqr(Params.H) - R2, 3.0f);
				}
			}
		}

		P->Rho[i] = Rho;
		P->P[i] = Params.GasConst * (Rho - Params.RestDensity);
	}
}

__global__ void ComputeForces(Parameters Params, Particles* P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P->NumP) {
		float3 FPress = make_float3(0.0f, 0.0f, 0.0f);
		float3 FVisc = make_float3(0.0f, 0.0f, 0.0f);

		for (int k = 0; k < 27; k++) {
			// Cell is between 0 and NumCells
			if ((P->NeighbourCells[i * 27 + k] < P->NumCells && P->NeighbourCells[i * 27 + k] >= 0)) {
				continue;
			}


			int CellStart = P->Cells[P->NeighbourCells[i * 27 + k] * 2];
			int CellEnd = CellStart + P->Cells[P->NeighbourCells[i * 27 + k] * 2 + 1] - 1;

			// Number of particles in cell is atleast 1
			if (CellEnd < CellStart) {
				continue;
			}

			for (int j = CellStart; j <= CellEnd; j++) {
				if (i == j) {
					continue;
				}

				float3 Rij;
				Rij = P->X[j] - P->X[i];
				float R2 = Sqr(Rij);
				float R = sqrtf(R2);

				if (R < Params.H) {
					float3 RNorm = Rij / R;

					float Pij = P->P[i] + P->P[j];

					FPress = FPress + ((RNorm * -1.0f) * Params.Mass * Pij / (2.0f * P->Rho[j]) * SpikyGrad(Params.H) * powf(Params.H - R, 3.0f));

					float3 Vij = P->V[j] - P->V[i];

					FVisc = FVisc + (Params.Visc * Params.Mass * Vij / P->Rho[j] * ViscLap(Params.H) * (Params.H - R));
				}
			}
		}

		float3 FGrav = Params.G * Params.Mass / P->Rho[i];

		P->F[i] = FPress + FVisc + FGrav;
	}
}

__global__ void UpdateVelocityHalf(Parameters Params, Particles* P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P->NumP) {
		P->V[i] = P->V[i] + ((Params.DT / 2.0f) * P->F[i] / P->Rho[i]);
	}
}

__global__ void UpdatePosition(Parameters Params, Particles* P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P->NumP) {
		P->X[i] = P->X[i] + (Params.DT * P->V[i]);
	}
}

__global__ void ComputeBoundries(Parameters Params, Particles* P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < P->NumP) {
		if (P->X[i].x - Params.H < 0.0f) {
			P->V[i].x = P->V[i].x * Params.BoundDamp;
			P->X[i].x = Params.H;
		}
		if (P->X[i].x + Params.H > Params.DomainX) {
			P->V[i].x = P->V[i].x * Params.BoundDamp;
			P->X[i].x = Params.DomainX - Params.H;
		}

		if (P->X[i].y - Params.H < 0.0f) {
			P->V[i].y = P->V[i].y * Params.BoundDamp;
			P->X[i].y = Params.H;
		}
		if (P->X[i].y + Params.H > Params.DomainY) {
			P->V[i].y = P->V[i].y * Params.BoundDamp;
			P->X[i].y = Params.DomainY - Params.H;
		}

		if (P->X[i].z - Params.H < 0.0f) {
			P->V[i].z = P->V[i].z * Params.BoundDamp;
			P->X[i].z = Params.H;
		}
		if (P->X[i].z + Params.H > Params.DomainZ) {
			P->V[i].z = P->V[i].z * Params.BoundDamp;
			P->X[i].z = Params.DomainZ - Params.H;
		}
	}
}

__global__ void FindParticleCellIDs(Parameters Params, Particles* P) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < P->NumP) {
		int CellID = CalculateParticleCellID(Params, P->X[i]);
		P->ParticleCellID[i] = CellID;
	}
}

__global__ void SetIndices(int* Indices, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		Indices[i] = i;
	}
}

__global__ void SortParticleData(int* Indices, Particles* P, float3* TX, float3* TV, float3* TF, float* TP, float* TRho) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < P->NumP) {
		TX[i] = P->X[i];
		TV[i] = P->V[i];
		TF[i] = P->F[i];
		TP[i] = P->P[i];
		TRho[i] = P->Rho[i];
	}
}

__global__ void CreateCells(int* Histogram, int* PrefixSum, int* Cells, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		Cells[i * 2] = PrefixSum[i];
		Cells[i * 2 + 1] = Histogram[i];
	}
}

void PutParticlesInCells(Parameters Params, Particles* P) {
	int BlockSize = 256;
	int NumBlocks = (P->NumP + BlockSize - 1) / BlockSize;

	FindParticleCellIDs << <NumBlocks, BlockSize >> > (Params, P);

	int* ParticleCellID_Out;
	int* Indices_In;
	int* Indices_Out;

	hipMalloc(&ParticleCellID_Out, sizeof(int) * P->NumP);
	hipMalloc(&Indices_In, sizeof(int) * P->NumP);
	hipMalloc(&Indices_Out, sizeof(int) * P->NumP);

	SetIndices << <NumBlocks, BlockSize >> > (Indices_In, P->NumP);

	hipDeviceSynchronize();

	void* TempStorage = NULL;
	size_t TempStorageBytes = 0;

	hipcub::DeviceRadixSort::SortPairs(TempStorage, TempStorageBytes, P->ParticleCellID, ParticleCellID_Out, Indices_In, Indices_Out, P->NumP);

	hipMalloc(&TempStorage, TempStorageBytes);

	hipcub::DeviceRadixSort::SortPairs(TempStorage, TempStorageBytes, P->ParticleCellID, ParticleCellID_Out, Indices_In, Indices_Out, P->NumP);

	hipDeviceSynchronize();
	
	hipFree(TempStorage);

	hipMemcpy(P->ParticleCellID, ParticleCellID_Out, sizeof(int) * P->NumP, hipMemcpyDeviceToDevice);
	
	hipFree(ParticleCellID_Out);
	hipFree(Indices_In);

	float3* TX;
	float3* TV;
	float3* TF;
	float* TP;
	float* TRho;

	hipMalloc(&TX, sizeof(float3) * P->NumP);
	hipMalloc(&TV, sizeof(float3) * P->NumP);
	hipMalloc(&TF, sizeof(float3) * P->NumP);
	hipMalloc(&TP, sizeof(float) * P->NumP);
	hipMalloc(&TRho, sizeof(float) * P->NumP);

	SortParticleData << <NumBlocks, BlockSize >> > (Indices_Out, P, TX, TV, TF, TP, TRho);
	hipDeviceSynchronize();

	hipMemcpy(P->X, TX, sizeof(float3) * P->NumP, hipMemcpyDeviceToDevice);
	hipMemcpy(P->V, TV, sizeof(float3) * P->NumP, hipMemcpyDeviceToDevice);
	hipMemcpy(P->F, TF, sizeof(float3) * P->NumP, hipMemcpyDeviceToDevice);
	hipMemcpy(P->P, TP, sizeof(float) * P->NumP, hipMemcpyDeviceToDevice);
	hipMemcpy(P->Rho, TRho, sizeof(float) * P->NumP, hipMemcpyDeviceToDevice);

	hipFree(TX);
	hipFree(TV);
	hipFree(TF);
	hipFree(TP);
	hipFree(TRho);

	hipFree(Indices_Out);

	int* Histogram;
	hipMalloc(&Histogram, sizeof(int) * P->NumCells);

	TempStorage = NULL;
	TempStorageBytes = 0;

	hipcub::DeviceHistogram::HistogramEven(TempStorage, TempStorageBytes, P->ParticleCellID, Histogram, P->NumCells, 0, P->NumCells, P->NumP);

	hipMalloc(&TempStorage, TempStorageBytes);

	hipcub::DeviceHistogram::HistogramEven(TempStorage, TempStorageBytes, P->ParticleCellID, Histogram, P->NumCells, 0, P->NumCells, P->NumP);

	hipDeviceSynchronize();

	hipFree(TempStorage);

	int* PrefixSum;

	hipMalloc(&PrefixSum, sizeof(int) * P->NumCells);

	TempStorage = NULL;
	TempStorageBytes = 0;

	hipcub::DeviceScan::ExclusiveSum(TempStorage, TempStorageBytes, Histogram, PrefixSum, P->NumCells);

	hipMalloc(&TempStorage, TempStorageBytes);

	hipcub::DeviceScan::ExclusiveSum(TempStorage, TempStorageBytes, Histogram, PrefixSum, P->NumCells);
	hipDeviceSynchronize();

	hipFree(TempStorage);

	NumBlocks = (P->NumCells + BlockSize - 1) / BlockSize;

	CreateCells << <NumBlocks, BlockSize >> > (Histogram, PrefixSum, P->Cells, P->NumCells);
	hipDeviceSynchronize();

	hipFree(Histogram);
	hipFree(PrefixSum);
}

Particles* allocateMemory(int NumP, int NumCells) {
	Particles* P;
	hipMallocManaged(&P, sizeof(Particles));
	hipMallocManaged(&(P->X), sizeof(float3) * NumP);
	hipMallocManaged(&(P->F), sizeof(float3) * NumP);
	hipMallocManaged(&(P->V), sizeof(float3) * NumP);
	hipMallocManaged(&(P->P), sizeof(float) * NumP);
	hipMallocManaged(&(P->Rho), sizeof(float) * NumP);
	hipMallocManaged(&(P->Cells), sizeof(int) * NumCells);
	hipMallocManaged(&(P->NeighbourCells), sizeof(int) * NumP * 27);
	hipMallocManaged(&(P->ParticleCellID), sizeof(float) * NumP);

	P->NumP = NumP;
	P->NumCells = NumCells;

	return P;
}

void InitSph(Parameters Params, Particles* P) {
	for (int i = 0; i < P->NumP; i++) {
		P->V[i] = make_float3(0.0f, 0.0f, 0.0f);
		P->F[i] = make_float3(0.0f, 0.0f, 0.0f);
		P->P[i] = 0.0f;
		P->Rho[i] = 0.0f;
	}

	for (int i = 0; i < P->NumCells; i++) {
		P->Cells[i] = -1;
	}

	int nParticles = 0;

	for (float x = Params.DomainX / 2 - Params.DomainX / 4; x <= Params.DomainX / 2 + Params.DomainX / 4; x += Params.H) {
		for (float y = Params.DomainY / 2 - Params.DomainY / 4; y <= Params.DomainY / 2 + Params.DomainY / 4; y += Params.H) {
			for (float z = Params.H; z < Params.DomainZ - Params.H; z += Params.H) {
				if (nParticles < P->NumP) {
					float jitter = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
					P->X[nParticles].x = x + jitter;
					P->X[nParticles].y = y + jitter;
					P->X[nParticles].z = z;
					nParticles++;
				}
				else {
					return;
				}
			}
		}
	}
}

void FirstStep(Parameters Params, Particles* P) {
	int BlockSize = 256;
	int NumBlocks = (P->NumP + BlockSize - 1) / BlockSize;

	CalculateNeighbours << <NumBlocks, BlockSize >> > (Params, P);
	ComputeDensityPressure << <NumBlocks, BlockSize >> > (Params, P);
	ComputeForces << <NumBlocks, BlockSize >> > (Params, P);
	UpdateVelocityHalf << <NumBlocks, BlockSize >> > (Params, P);
	//UpdateVelocityHalf << <NumBlocks, BlockSize >> > (Params, P);
	UpdatePosition << <NumBlocks, BlockSize >> > (Params, P);
	ComputeBoundries << <NumBlocks, BlockSize >> > (Params, P);
	hipDeviceSynchronize();
}

void SimStep(Parameters Params, Particles* P) {
	int BlockSize = 256;
	int NumBlocks = (P->NumP + BlockSize - 1) / BlockSize;

	CalculateNeighbours << <NumBlocks, BlockSize >> > (Params, P);
	ComputeDensityPressure << <NumBlocks, BlockSize >> > (Params, P);
	ComputeForces << <NumBlocks, BlockSize >> > (Params, P);
	UpdateVelocityHalf << <NumBlocks, BlockSize >> > (Params, P);
	UpdateVelocityHalf << <NumBlocks, BlockSize >> > (Params, P);
	UpdatePosition << <NumBlocks, BlockSize >> > (Params, P);
	ComputeBoundries << <NumBlocks, BlockSize >> > (Params, P);
	hipDeviceSynchronize();
}